#include "hip/hip_runtime.h"
#include "fft.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

//-----------------------------------------------------------------------------
__global__ void kernelFunc(float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d, const unsigned int N, const unsigned int M, int j) 
{
	int k = bx * 512 + tx;
	
	int m, n;
	float z_r, z_i, w_r, w_i, temp_r, temp_i;
	
	if(j == N/2) {
		int i, p, q;
		
		p = 2*k;
		q = 0;
		
		for(i=0; i<M; i++) {
			q = q + ((p & (1 << i)) >> i) * (1 << (M-i-1));
		}
		
		X_r_d[p] = x_r_d[q];
		X_i_d[p] = x_i_d[q];
		X_r_d[p+1] = x_r_d[q+N/2];
		X_i_d[p+1] = x_i_d[q+N/2];
	}
		
		m = (k*j)%(N/2);
		n = (k/(N/(2*j)))*(N/j) + (k%(N/(2*j)));
		
		w_r =  cos((2*PI*m)/N);
		w_i = -sin((2*PI*m)/N);
		
		z_r = w_r*X_r_d[n+N/(2*j)] - w_i*X_i_d[n+N/(2*j)];
		z_i = w_r*X_i_d[n+N/(2*j)] + w_i*X_r_d[n+N/(2*j)];
		
		temp_r = X_r_d[n];
		temp_i = X_i_d[n];
		
		X_r_d[n] = temp_r + z_r;
		X_i_d[n] = temp_i + z_i;
		X_r_d[n+N/(2*j)] = temp_r - z_r;
		X_i_d[n+N/(2*j)] = temp_i - z_i;
	
}
//-----------------------------------------------------------------------------
__global__ void kernelFunc2(float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d, const unsigned int N, const unsigned int M, int j) 
{
	int k = bx * 512 + tx;
	
	if(j == N/2) {
		
		int i, p, q;
			
		p = 2*k;
		q = (p & 1) * N/2;
		
		for(i=1; i<M; i+=2) {
			q = q + ((p & (3 << i)) >> i) * (1 << M-i-2); 
		}
		
		X_r_d[q] = x_r_d[p] + x_r_d[p+1];
		X_i_d[q] = x_i_d[p] + x_i_d[p+1];
		X_r_d[q+N/2] = x_r_d[p] - x_r_d[p+1];
		X_i_d[q+N/2] = x_i_d[p] - x_i_d[p+1];
	
	} else {
		int i, m, n;
		float z_r[4], z_i[4], w_r[4], w_i[4], temp_r[4], temp_i[4];

		n = (k/(N/(4*j)))*(N/j) + (k%(N/(4*j)));
		
		for(i=0; i<4; i++){
			
			temp_r[i] = x_r_d[n+i*N/(4*j)];
			temp_i[i] = x_i_d[n+i*N/(4*j)];
			
			m = (j==N/4)?0:(k%(N/(4*j)))*i*j;
			
			w_r[i] =  cos((2*PI*m)/N);
			w_i[i] = -sin((2*PI*m)/N);
			
		}

		z_r[0] = temp_r[0] + temp_r[1] + temp_r[2] + temp_r[3];
		z_i[0] = temp_i[0] + temp_i[1] + temp_i[2] + temp_i[3];

		z_r[1] = temp_r[0] + temp_i[1] - temp_r[2] - temp_i[3]; 
		z_i[1] = temp_i[0] - temp_r[1] - temp_i[2] + temp_r[3];
		
		z_r[2] = temp_r[0] - temp_r[1] + temp_r[2] - temp_r[3]; 
		z_i[2] = temp_i[0] - temp_i[1] + temp_i[2] - temp_i[3];
		
		z_r[3] = temp_r[0] - temp_i[1] - temp_r[2] + temp_i[3]; 
		z_i[3] = temp_i[0] + temp_r[1] - temp_i[2] - temp_r[3];
		
		for(i=0; i<4; i++){
			
			x_r_d[n+i*N/(4*j)] = w_r[i]*z_r[i] - w_i[i]*z_i[i];
			x_i_d[n+i*N/(4*j)] = w_r[i]*z_i[i] + w_i[i]*z_r[i];
			
		}
		
		if(j == N/4) {
			int p, q;
			
			p = 4*k;
			q = 0;
			
			for(i=0; i<M; i+=2) {
				q = q + ((p & (3 << i)) >> i) * (1 << M-i-2);
			}
			
			X_r_d[q] = x_r_d[p];
			X_i_d[q] = x_i_d[p];
			X_r_d[q+N/4] = x_r_d[p+1];
			X_i_d[q+N/4] = x_i_d[p+1];
			X_r_d[q+N/2] = x_r_d[p+2];
			X_i_d[q+N/2] = x_i_d[p+2];
			X_r_d[q+(3*N)/4] = x_r_d[p+3];
			X_i_d[q+(3*N)/4] = x_i_d[p+3];
		}
	}
}
//-----------------------------------------------------------------------------
void gpuKernel_simple(float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d, const unsigned int N, const unsigned int M)
{	
	int j;
	
	if(N <= 1024) {
		dim3 dimGrid(1,1);
		dim3 dimBlock(N/2,1);
		
		for(j=N/2; j>=1; j=j/2) {
			kernelFunc <<< dimGrid, dimBlock >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, j);
		}
	} else {
		dim3 dimGrid(N/1024,1);
		dim3 dimBlock(512,1);
		
		for(j=N/2; j>=1; j=j/2) {
			kernelFunc <<< dimGrid, dimBlock >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, j);
		}
	}	
}
//-----------------------------------------------------------------------------
void gpuKernel_efficient(float* x_r_d, float* x_i_d, float* X_r_d, float* X_i_d, const unsigned int N, const unsigned int M)
{
	int j;

	dim3 dimGrid(N/2048,1);
	dim3 dimBlock(512,1);

	for(j=1; j<N/2; j*=4) {
		kernelFunc2 <<< dimGrid, dimBlock >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, j);
	}
	
	if(M % 2 == 1) {
		dim3 dimGrid2(N/1024,1);
		dim3 dimBlock2(512,1);
	
		kernelFunc2 <<< dimGrid2, dimBlock2 >>>(x_r_d, x_i_d, X_r_d, X_i_d, N, M, N/2);
	}
	
}
